/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either ex  ess or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmark/benchmark.h>

#include <hip/hip_runtime_api.h>
#include <rmm/thrust_rmm_allocator.h>
#include <rmm/device_uvector.hpp>
#include "rmm/mr/device/cnmem_memory_resource.hpp"
#include "rmm/mr/device/default_memory_resource.hpp"

static void BM_UvectorSizeConstruction(benchmark::State& state)
{
  rmm::mr::cnmem_memory_resource mr{};
  rmm::mr::set_default_resource(&mr);

  for (auto _ : state) {
    rmm::device_uvector<int32_t>(state.range(0), rmm::stream_t{});
    hipDeviceSynchronize();
  }
}
BENCHMARK(BM_UvectorSizeConstruction)
  ->RangeMultiplier(10)
  ->Range(10'000, 1'000'000'000)
  ->Unit(benchmark::kMicrosecond);

static void BM_ThrustVectorSizeConstruction(benchmark::State& state)
{
  rmm::mr::cnmem_memory_resource mr{};
  rmm::mr::set_default_resource(&mr);
  for (auto _ : state) {
    rmm::device_vector<int32_t>(state.range(0));
    hipDeviceSynchronize();
  }
}

BENCHMARK(BM_ThrustVectorSizeConstruction)
  ->RangeMultiplier(10)
  ->Range(10'000, 1'000'000'000)
  ->Unit(benchmark::kMicrosecond);

BENCHMARK_MAIN();
